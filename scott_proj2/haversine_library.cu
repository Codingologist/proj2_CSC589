#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void haversine_distance_kernel(int size, const double *x1, const double *y1,
                                          const double *x2, const double *y2, double *dist)
{
    // use any references to compute haversine distance bewtween (x1,y1) and (x2,y2), given in vectors/arrays
    // e.g., https://stackoverflow.com/questions/27928/calculate-distance-between-two-latitude-longitude-points-haversine-formula
    int ind = blockIdx.x * blockDim.x + threadIdx.x; // block index
    int R = 6371;

    if (ind < size)
    {
        double dLat = (y2[ind] - y1[ind]) * (M_PI / 180);
        double dLon = (x2[ind] - x1[ind]) * (M_PI / 180);
        double y1Rad = y1[ind] * (M_PI / 180);
        double y2Rad = y2[ind] * (M_PI / 180);

        double a = sin(dLat / 2) * sin(dLat / 2) + cos(y1Rad) * cos(y2Rad) * sin(dLon / 2) * sin(dLon / 2);
        double c = 2 * atan2(sqrt(a), sqrt(1 - a));
        dist[ind] = R * c;
    }
}

void run_kernel(int size, const double *x1, const double *y1, const double *x2, const double *y2, double *dist)

{
    dim3 dimBlock(1024);
    printf("in run_kernel dimBlock.x=%d\n", dimBlock.x);

    dim3 dimGrid(ceil((double)size / dimBlock.x));

    haversine_distance_kernel<<<dimGrid, dimBlock>>>(size, x1, y1, x2, y2, dist);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::stringstream strstr;
        strstr << "run_kernel launch failed" << std::endl;
        strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
        strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
        strstr << hipGetErrorString(error);
        throw strstr.str();
    }
}